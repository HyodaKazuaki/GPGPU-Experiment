#define HANDLE_ERROR(err) if(err != hipSuccess) { printf("Error\n"); exit(1); }

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 32

__global__ void add(int *a, int *b, int *c){
    int tid = blockIdx.x;
    if(tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(int argc, char *argv[]){
    int num_gpu = 0;
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    if(argc == 2) num_gpu = atoi(argv[1]);

    for (int i = 0; i< N; i++){
        a[i] = i;
        b[i] = i * i;
    }

    hipSetDevice(num_gpu);

    HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add <<< N, 1 >>> (dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
    for(int i = 0; i < N; i++)
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    return 0;
}
